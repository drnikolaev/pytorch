#include "hip/hip_runtime.h"
// Indexing tensors by by tensors
//
// This corresponds to "advanced indexing" in NumPy. The two operations are:
//
//  index(Tensor self, indices) -> Tensor
//  index_put_(Tensor self, indices, value, accumulate=false)
//
// The index is a TensorList containg kLong or kByte tensors or nulls. Byte
// tensors (boolean masks) are expanded to long tensors via nonzero(). Null
// tensors signify that the dimension is not indexed.
//
// All indexes are broadcast together and iterated as *one*. From NumPy:
//
// result[i_1, ..., i_M] == x[ind_1[i_1, ..., i_M], ind_2[i_1, ..., i_M],
//                           ..., ind_N[i_1, ..., i_M]]
//
// Note 1: ByteTensors expand to index as many dimensions as there are in the
// mask.
//
// Note 2: The behavior is more complicated when the index tensors are not all
// adjacent (e.g. x[[0, 1], :, [2, 3]]). In this case, self and the index
// tensors are transposed to the front: x.transpose(1, 2)[[0, 1], [2, 3]]
//
// The code contains two implementations of indexing. The more efficient
// implementation treats indexing like an elementwise operation over the
// tensors `result`, `x`, `ind_1`, `ind_2`, etc. This implementation does
// not work for index_put_ with accumulate=True. The other implementation
// combines the indexed tensors into a single linear index that is used
// with Tensor.put_. This is used for index_put_ with accumulate=True.
//
// The more efficient implementation takes the following steps for the
// above operation:
//
// 1) Broadcast ind_1, ind_2, ind_3 together to a common shape
// 2) Record x.stride(i) for each indexed dimension `i`
// 3) Replace the indexed subspace of `x` with the shape of the corresponding
//    subspace of `result` but with stride 0
// 4) Add dimensions of size 1 to the index tensors (ind_1, ind_2, etc.) so
//    that their shape is compatible with the result shape
//
// The CPU or CUDA kernel then computes element-wise over the broadcasted
// and restrided result, x, ind_1,  ind_2, etc.:
//
//   result[...] = *(&x[...] +
//                   ind_1[...] * x.stride(1) +
//                   ind_2[...] * x.stride(2) +
//                   ...)
//
// where & and * represent the C-style address-of and indirection operations.

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/ExpandUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Indexing.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <c10/core/ScalarType.h>

#include <ATen/native/Indexing.h>
#include <ATen/NativeFunctions.h>
#include <ATen/LegacyTHFunctions.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <hip/hip_vector_types.h>

#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <torch/csrc/utils/tensor_flatten.h>

#include <ATen/cpu/vec256/vec256.h>
#include <>

#ifdef __HIP_PLATFORM_HCC__
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif
#define GRID_SIZE 512
#define GROUP_SIZE 32
//#define BLOCK_SIZE 32

namespace at {
namespace native {

// FIXME
// MOVE THIS STUFF TO SOME COMMON PLACE SHARED BY .cpp and .cu

// STUFF BEGIN
[[noreturn]]
static void invalid_mask(const Tensor& self, int64_t idx, const Tensor& mask, int64_t maskIdx) {
  std::stringstream ss;
  ss << "The shape of the mask " << mask.sizes() << " at index " << maskIdx;
  ss << " does not match the shape of the indexed tensor " << self.sizes();
  ss << " at index " << idx;
  AT_INDEX_ERROR(ss.str());
}

static void checkIndexTensorTypes(TensorList indices) {
  for (auto& tensor : indices) {
    if (tensor.defined()) {
      auto scalarType = tensor.scalar_type();
      if (scalarType != kLong && scalarType != kByte) {
        AT_INDEX_ERROR("tensors used as indices must be long or byte tensors");
      }
    }
  }
}

static std::vector<Tensor> expandByteTensors(const Tensor& self, TensorList indices) {
  // Expands byte tensors (masks) into the equivalent indexing by LongTensors
  std::vector<Tensor> result;
  for (auto& index : indices) {
    if (index.scalar_type() == kByte) {
      // The sizes of the ByteTensor mask must match the sizes of the
      // corresponding dimensions in self
      for (int64_t j = 0; j < index.dim(); j++) {
        int64_t srcIdx = result.size() + j;
        if (index.size(j) != self.size(srcIdx)) {
          invalid_mask(self, srcIdx, index, j);
        }
      }
      // Replace with nonzeros
      auto nonzero = index.nonzero();
      auto special_empty = false;
      for (int64_t j = 0; j < index.dim(); j++) {
        if (special_empty) {
          // We can't call select on an empty tensor so we just create an empty
          // tensor.
          result.emplace_back(at::empty({0}, nonzero.options()));
        } else {
          result.emplace_back(nonzero.select(1, j));
        }
      }
    } else {
      result.emplace_back(index);
    }
  }
  return result;
}

static bool hasContiguousSubspace(TensorList tl) {
  // true if all the non-null tensors are adjacent
  auto isDefined = [](const Tensor& tensor) { return tensor.defined(); };
  auto isNull = [](const Tensor& tensor) { return !tensor.defined(); };
  auto start = std::find_if(tl.begin(), tl.end(), isDefined);
  auto stop = std::find_if(tl.rbegin(), tl.rend(), isDefined);
  auto it = std::find_if(start, stop.base(), isNull);
  return it == stop.base();
}

// Transposes the tensor and indices together so that all the non-null indices
// index the first k dimensions of the tensor. Returns the transposed tensor
// and the reordered indices. For example:
//  transposeToFront(tensor, {nullptr, a, nullptr, b})
// returns
//  tensor.permute([1, 3, 0, 2]), {a, b, nullptr, nullptr}
static std::tuple<Tensor, std::vector<Tensor>> transposeToFront(Tensor self, TensorList indices) {
  std::vector<int64_t> dims;
  std::vector<Tensor> transposedIndices;
  dims.reserve(self.dim());
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back(indices[i]);
    }
  }
  for (int64_t i = 0; i < self.dim(); i++) {
    if (!indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back();
    }
  }
  return std::make_tuple(self.permute(dims), std::move(transposedIndices));
}

static std::vector<int64_t> computeLinearStride(const Tensor& tensor) {
  // computes the stride as if tensor were contigous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1,
      std::multiplies<int64_t>());
  return stride;
}

// Unsqueezes src `before` times at the front and `after` times at the end
static Tensor unsqueezeN(const Tensor& src, int64_t before, int64_t after) {
  auto srcSizes = src.sizes();
  auto nDim = src.dim();
  std::vector<int64_t> sizes(nDim + before + after, 1);
  for (int64_t i = 0; i < nDim; i++) {
    sizes[i + before] = srcSizes[i];
  }
  return src.view(sizes);
}

// STUFF END


__global__ void arange_kernel(int64_t n, int64_t* a) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
      i += blockDim.x * gridDim.x) {
    a[i] = i;
  }
}

static std::tuple<Tensor, Tensor, Tensor, Tensor, int64_t, int64_t, int64_t, int64_t>
makeLinearIndex(Tensor self, TensorList orig) {
  checkIndexTensorTypes(orig);
  // first expand ByteTensor (boolean masks) into 1 or more LongTensors
  auto indices = expandByteTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t) self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices) = transposeToFront(self, indices);
  }

  auto strides = computeLinearStride(self);

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1;
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      Tensor index = (indices[i].remainder(self.size(i)) * strides[i]).to(kLong);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= self.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= self.size(i);
    }
  }

  // Compute the linear indices for the parts of the tensor not being indexed
  // ...and not being sorted
  Tensor beforeIndex;
  if (emptyBefore > 0) {
    Tensor index = at::native::empty_cuda({nElemBefore},
        self.options().dtype(kLong).device(at::DeviceType::CUDA));
    int64_t *pData = index.data<int64_t>();
    void* args[] = {&nElemBefore, &pData};
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    THCudaCheck(
        hipLaunchKernel((const void*) &arange_kernel,
            WARP_SIZE, GRID_SIZE, args, 0, stream));
    THCudaCheck(hipStreamSynchronize(stream));
    index = index * strides[emptyBefore - 1];
    index = index.view(self.sizes().slice(0, emptyBefore));
    beforeIndex = unsqueezeN(index, 0, linearIndex.dim() + emptyAfter);
  }
  Tensor afterIndex;
  if (emptyAfter > 0) {
    Tensor index = at::native::empty_cuda({nElemAfter},
        self.options().dtype(kLong).device(at::DeviceType::CUDA));
    int64_t *pData = index.data<int64_t>();
    void* args[] = {&nElemAfter, &pData};
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    THCudaCheck(
        hipLaunchKernel((const void*) &arange_kernel,
            WARP_SIZE, GRID_SIZE, args, 0, stream));
    THCudaCheck(hipStreamSynchronize(stream));
    index = index.view(self.sizes().slice(self.dim() - emptyAfter, emptyAfter));
    afterIndex = unsqueezeN(index, linearIndex.dim() + emptyBefore, 0);
  }

  return std::make_tuple(self, linearIndex, beforeIndex, afterIndex, emptyBefore, emptyAfter,
      nElemBefore, nElemAfter);
}

template<typename IndexType>
__device__ __forceinline__
IndexType indexToOffset(IndexType dims, const IndexType* sizes, const IndexType* strides,
    IndexType linearIndex) {
  IndexType offset(0);
  for (IndexType i = dims - 1; i > 0; --i) {
    offset += (linearIndex % sizes[i]) * strides[i];
    linearIndex /= sizes[i];
  }
  return offset + linearIndex* strides[0];
}

template<typename index_t>
__device__ __forceinline__
index_t extended_idx(index_t nseq, index_t sortedSize,
    index_t extendedSize, index_t extendedStride, const int64_t* origOrder) {
  index_t aPart = nseq % sortedSize;
  index_t aPartOrdered = origOrder[aPart];
  index_t ret = nseq + (aPartOrdered - aPart) * extendedStride;
  if (ret < 0) {
    ret += extendedSize;
  } else if (ret >= extendedSize) {
    ret -= extendedSize;
  }
//printf("============== %d %d      %d   %d * %d --> %d --> %d\n", nseq, aPart, aPartOrdered,
//sortedSize, extendedStride, nseq + (aPartOrdered - aPart) * extendedStride, ret);
  return ret;
}

//template<typename index_t>
//__device__ __forceinline__
//index_t extended_pos(index_t nseq, index_t sortedSize,
//    index_t extendedStride, const int64_t* origOrder, const int64_t* extendedIdx) {
//  return extendedIdx[extended_idx(nseq, sortedSize, extendedStride, origOrder)];
//}


template<typename scalar_t>
__global__
void backward_indexing_sum(scalar_t* dstData, int64_t* offsets, const scalar_t* values,
    int64_t extendedSize, int64_t extendedStride) {
  using accscalar_t = acc_type<scalar_t, true>;
  int actualGroupSize = extendedSize / extendedStride;
  int blockSize = blockDim.x * blockDim.y * blockDim.z;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < extendedStride;// * GROUP_SIZE;
       i += blockDim.x * gridDim.x) {
    int beg = i % extendedStride;
    for (int t = beg; t < beg + actualGroupSize; ++t) {
      dstData[offsets[t]] += values[t];
    }
  }
}


template<typename scalar_t>
__global__
void backward_indexing_kernel(const int64_t* extendedIdx,
    const int64_t* origOrder, const scalar_t* gradValues,
    int64_t extendedSize, int64_t extendedStride, int64_t sortedSize,
    scalar_t* dstData, int64_t dstDims,
    const int64_t* dstSizes, const int64_t* dstStrides) { //, int64_t* offsets, scalar_t* values) {
  using accscalar_t = acc_type<scalar_t, true>;

  int blockSize = blockDim.x * blockDim.y * blockDim.z;
//  int idxMax = (extendedSize + blockSize - 1) / blockSize;
  int idx = blockIdx.x;
//  int idy = blockIdx.y;

//  __shared__ int orderArr[GROUP_SIZE];
  __shared__ int offsetArr[GROUP_SIZE];
  __shared__ accscalar_t valArr[GROUP_SIZE];

//  int blockHeadPos = extended_pos<int>(idx * blockSize, sortedSize,
//      extendedStride, origOrder, extendedIdx);
//  printf("!!!!!!!!!!!! %d %d %d %d\n", idxMax, idx, idy, blockSize);

  const int realGroup = extendedSize / extendedStride;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < extendedSize;
      i += blockDim.x * gridDim.x) {
//    __syncwarp(GROUP_SIZE - 1);
    __syncthreads();
//    const int currGroup = i % realGroup;
    const int j = (i % realGroup) * extendedStride + i / realGroup;  // j goes along sorted dimensions

//    const int g = i % realGroup;
//    const int currEntry = i;  // all duplicates in one bucket
//    const int currStride = currEntry / extendedStride;

    const int th = i % GROUP_SIZE;
    const int entriesDone = idx * GROUP_SIZE;
    const int extIdx = extended_idx<int>(j, sortedSize, extendedSize, extendedStride, origOrder);
    const int dstIdx = extendedIdx[extIdx];

//printf("################### %d %d %d     %d + %d: %d  \n", idx,i,j,entriesDone, th,  extIdx);

    offsetArr[th] = indexToOffset<int64_t>(dstDims, dstSizes, dstStrides,
        dstIdx);

//        offsetArr[th] = extendedIdx[extIdx]; //offset;
        valArr[th] = gradValues[extIdx];

//const int    offset = extendedIdx[extIdx]; //offset;
//const scalar_t value = gradValues[extIdx];

//    printf("%d %lld      %d %lld  \n", i, offsets[i], j, offsets[j]);
//
//
      __syncthreads();

    //      const int g = i % realGroup;
      if (th == 0) {
//        int tl = realGroup < GROUP_SIZE ? realGroup : GROUP_SIZE;
        int currentBlockEnd = extendedSize - entriesDone;
          #pragma unroll
          for (int t = 0; t < GROUP_SIZE; ++t ) {
            if (t >= currentBlockEnd) {
              break;
            }
//            printf("%d %d %lld  %lld\n", g, t, g*extendedStride+t, offsets[g*extendedStride+t]);

            //          offsets[i] = extendedIdx[extIdx]; //offset;
            //          values[i] = gradValues[extIdx];

  //          int jt = orderArr[t];
            dstData[offsetArr[t]] += valArr[t];

           printf("%d      %d  \n", t, offsetArr[t]);


          }
      }
  }



/*
  __shared__ int offsetArr[GROUP_SIZE];
  __shared__ accscalar_t valArr[GROUP_SIZE];

  offsetArr[th] = extendedIdx[no];
  valArr[th] = gradValues[no];
  __syncthreads();
  if (th == 0 && idx < idxMax) {
    int currentBlockEnd = extendedIdxSize - idx * GROUP_SIZE;
    currentBlockEnd =
        currentBlockEnd < GROUP_SIZE ? currentBlockEnd : GROUP_SIZE;

    //     printf("                %d %d %d %d\n", idx, i, th, currentBlockEnd); // TODO!!!

    for (int t = 0; t < currentBlockEnd; ++t) {
      dstData[offsetArr[t]] += valArr[t];

      //        printf("%d %d %d\n", idx, t, offsetArr[t]);
    }
    __threadfence();
  }
*/


  /*
      if (th == 0) {
        int currentBlockEnd = extendedSize - idx * GROUP_SIZE;
        currentBlockEnd = currentBlockEnd < GROUP_SIZE ? currentBlockEnd : GROUP_SIZE;

        int beg = i % extendedStride;
        for (int t = beg; t < beg + GROUP_SIZE; ++t) {
          offsets[t] = offsetArr[t-beg];
          values[t] = valArr[t-beg];
        }

        for (int t = 0; t < GROUP_SIZE; ++t) {
          dstData[offsetArr[t]] += valArr[t];
          __threadfence();
        }
      }
    */
      //ChildKernel<<<1, 32>>>(data);
      //hipDeviceSynchronize();

//      ++idx;
//      if (idx >= idxMax) {
//        ///        printf("***************** %d %d %d\n", blockHeadPos, idx, blockSize);
//        break;
//      }
//
//      int blockHeadPosNext = extended_pos<int>(idx * blockSize + extendedStride, sortedSize,
//          extendedStride, origOrder, extendedIdx);
//      if (blockHeadPosNext != blockHeadPos) {
//        // next block is processing other images' indexes, exit.
//        printf("---- %d != %d  %d %d\n", blockHeadPosNext, blockHeadPos, idx, blockSize);
//        break;
//      }
//      blockHeadPos = blockHeadPosNext; // keep going in the same warp
//
//      printf("&&&&&&&&&&& %d %d %d\n", blockHeadPos, idx, blockSize);
//
//    } else {
//
//      printf("XXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXXX %d %d %d\n", blockHeadPos, idx, blockSize);
//
//    }


//  }
}

template<typename T>
struct TensorAccumFullyIndexedPutOp : thrust::unary_function<int64_t, T> {
  TensorAccumFullyIndexedPutOp(at::cuda::detail::TensorInfo<T, int64_t> info, const T* psrc,
      const int64_t* sorted_idx_beg, const int64_t* sorted_idx_end, const int64_t* orig_idx_beg)
      : info(info), p_source(psrc), sorted_idx_beg(sorted_idx_beg), sorted_idx_end(sorted_idx_end),
        orig_idx_beg(orig_idx_beg) {}

  __device__ __forceinline__ T
  operator()(int64_t n) {
    const int64_t no = orig_idx_beg[n]; // restore if flipped
    const T* pvalue = p_source + no;
    const int64_t* pindex = sorted_idx_beg + no;
    const int64_t linear_index = *pindex;
    const int64_t offset = indexToOffset<int64_t>(info.dims, info.sizes, info.strides,
        linear_index);

    if (pindex == sorted_idx_beg || *pindex != *(pindex - 1)) {
      do {
        info.data[offset] += *pvalue;
        pindex++;
        pvalue++;
      } while (pindex != sorted_idx_end && *pindex == linear_index);
    }
    return 0; // discarded
  }

  const at::cuda::detail::TensorInfo<T, int64_t> info;
  const T* p_source;
  const int64_t* sorted_idx_beg;
  const int64_t* sorted_idx_end;
  const int64_t* orig_idx_beg;
};



long cnt = 0L;
long total = 0L;





Tensor& index_put_cuda_(Tensor& self_, TensorList indices, const Tensor& value,
    bool accumulate) {
  if (indices.size() > (size_t) self_.dim()) {
    AT_INDEX_ERROR("too many indices for tensor of dimension ", self_.dim(), " (got ",
        indices.size(), ")");
  }
  Tensor self, linearIndex;
  Tensor beforeIndex, afterIndex;
  int64_t emptyBefore = 0L, emptyAfter = 0L;
  int64_t nElemBefore = 1L, nElemAfter = 1L;

  std::tie(self, linearIndex, beforeIndex, afterIndex, emptyBefore, emptyAfter, nElemBefore,
      nElemAfter) = makeLinearIndex(self_, indices);

  auto sortedLinearIndex = linearIndex.clone();
  auto origCounters = at::empty_like(linearIndex);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  int64_t idxSize = linearIndex.numel(); // const breaks usin cudaKernelLaunch
  int64_t dstSize = self.numel();
  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "xput_cuda_1", [&] {
    int64_t* sortedLinearIndex_beg = sortedLinearIndex.data<int64_t>();
    int64_t* sortedLinearIndex_end = sortedLinearIndex_beg + idxSize;
    int64_t* origCounters_beg = origCounters.data<int64_t>();
    auto sortedLinearIndex_iter = thrust::device_ptr<int64_t>(sortedLinearIndex_beg);
    auto origCounters_iter = thrust::device_ptr<int64_t>(origCounters_beg);
    auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
    TensorAccumFullyIndexedPutOp<scalar_t> aiPutOp(self_info, value.data<scalar_t>(),
        sortedLinearIndex_beg, sortedLinearIndex_end, origCounters_beg);

//++cnt;
//auto start = std::chrono::high_resolution_clock::now();

    void* args[] = {&idxSize, &origCounters_beg};
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    THCudaCheck(
        hipLaunchKernel((const void*) &arange_kernel,
            WARP_SIZE, GRID_SIZE, args, 0, stream));
    THCudaCheck(hipStreamSynchronize(stream));


//    auto finish = std::chrono::high_resolution_clock::now();
//    total += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//    if (cnt % 100 == 0) {
//      std::cout << "seq: " << total / cnt << "ns" << std::endl;
//    }

    thrust::sort_by_key(policy, sortedLinearIndex_iter, sortedLinearIndex_iter + idxSize,
        origCounters_iter, ThrustLTOp<int64_t>());

    if (!beforeIndex.defined() && !afterIndex.defined()) {
      // Full size index, done:
      thrust::counting_iterator<int64_t> first(0);
      thrust::counting_iterator<int64_t> last(idxSize);
      thrust::for_each(policy, first, last, aiPutOp);
    }
  });

  if (beforeIndex.defined() || afterIndex.defined()) {
    // Sum with broadcasting to compute the full index
    // using unsorted original
    linearIndex = unsqueezeN(linearIndex, emptyBefore, emptyAfter);
    if (emptyBefore > 0) {
      linearIndex = linearIndex + beforeIndex;
    }
    if (emptyAfter > 0) {
      linearIndex = linearIndex + afterIndex;
    }

    std::cout << "### origCounters" << std::endl;
    print(origCounters, 120);
    std::cout << std::endl
              << "strides: " << computeLinearStride(origCounters)
              << std::endl
              << std::endl;
    std::cout << "### linearIndex" << std::endl;
    print(linearIndex, 120);
    std::cout << std::endl
              << "strides: " << computeLinearStride(linearIndex)
              << std::endl
              << std::endl;



    AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "index_put_cuda_kernel_", [&] {
      cuda::detail::TensorInfo <scalar_t, int64_t> self_info =
          cuda::detail::getTensorInfo<scalar_t, int64_t>(self);

//      cuda::detail::TensorInfo <int64_t, int64_t> sorted_info =
//          cuda::detail::getTensorInfo<int64_t, int64_t>(sortedLinearIndex);
//
      int64_t dstDims = self_info.dims;
//      int sortedDims = sorted_info.dims;
      Tensor dstSizes =
          CPU(kLong).tensorFromBlob(self_info.sizes, {dstDims}).cuda();
      Tensor dstStrides =
          CPU(kLong).tensorFromBlob(self_info.strides, {dstDims}).cuda();
//      Tensor sortedSizes =
//          CPU(kLong).tensorFromBlob(sorted_info.sizes, {sortedDims}).cuda();
//      Tensor sortedStrides =
//          CPU(kLong).tensorFromBlob(sorted_info.strides, {sortedDims}).cuda();
      int64_t* dstSizesPtr = dstSizes.data<int64_t>();
      int64_t* dstStridesPtr = dstStrides.data<int64_t>();
//      int64_t* sortedSizesPtr = sortedSizes.data<int64_t>();
//      int64_t* sortedStridesPtr = sortedStrides.data<int64_t>();

//      auto offsets = at::empty_like(linearIndex);
//      auto values = at::empty_like(self);
//      int64_t* offsetsPtr = offsets.data<int64_t>();
//      scalar_t* valuesPtr = values.data<scalar_t>();

      std::cout << "self" << std::endl;
      print(self, 120);

      std::cout << self.sizes() << std::endl
      << "strides: " << computeLinearStride(self)<< std::endl<< std::endl;
      std::cout << "value"  << std::endl;
      print(value, 120);
      std::cout << value.sizes() << std::endl
                << "strides: " << computeLinearStride(value)<< std::endl<< std::endl;

      scalar_t* valuePtr = value.data<scalar_t>();
      int64_t extendedSize = linearIndex.numel();
      int64_t* origCountersPtr = origCounters.data<int64_t>();
      int64_t* extendedLinearIndexPtr = linearIndex.data<int64_t>();

      dim3 gridSize(GRID_SIZE);
      dim3 blockSize(GROUP_SIZE);
      void* args[] = {&extendedLinearIndexPtr, &origCountersPtr, &valuePtr,
                      &extendedSize, &nElemAfter, &idxSize,
                      &self_info.data, &dstDims, &dstSizesPtr, &dstStridesPtr};//, &offsetsPtr, &valuesPtr};
//      void* args2[] = {&self_info.data, &offsetsPtr, &valuesPtr,
//                       &extendedSize, &nElemAfter};
      THCudaCheck(
          hipLaunchKernel((const void*) &backward_indexing_kernel<scalar_t>, gridSize, blockSize,
              args, 0, stream));
//      THCudaCheck(
//          hipLaunchKernel((const void*) &backward_indexing_sum<scalar_t>, gridSize, blockSize,
//              args2, 0, stream));

      THCudaCheck(hipStreamSynchronize(stream));



    });
  }

  return self_;
}

}}






//long cnt = 0L;
//long total = 0L;
//
//

//++cnt;
//auto start = std::chrono::high_resolution_clock::now();
//
//auto finish = std::chrono::high_resolution_clock::now();
//total += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//if (cnt % 100 == 0) {
//std::cout << "AFTER indices: " << total / cnt << "ns" << std::endl;
//}

//    std::cout << "afterIndex" << std::endl;
//                        print(afterIndex, 120);
//                        std::cout << std::endl
//                                  << "strides: " << computeLinearStride(afterIndex)
//                                  << std::endl
//                                  << std::endl;

//std::cout << "*********** beforeIndex" << std::endl;
//print(beforeIndex, 120);
//std::cout << beforeIndex.sizes() << std::endl
//<< "strides: " << computeLinearStride(beforeIndex)
//<< std::endl
//<< std::endl;
//
//std::cout << "linearIndex += before" << std::endl;
//print(linearIndex, 120);
//std::cout << linearIndex.sizes() << std::endl
//<< "strides: " << computeLinearStride(linearIndex)
//<< std::endl
//<< std::endl;

//std::cout << "self" << std::endl;
//print(self, 120);
//std::cout << self.sizes() << std::endl
//<< "strides: " << computeLinearStride(self)
//<< std::endl
//<< std::endl;
//
//std::cout << "origCounters" << std::endl;
//print(origCounters, 120);
//std::cout << origCounters.sizes() << std::endl
//<< "strides: " << computeLinearStride(origCounters)
//<< std::endl
//<< std::endl;

//      std::cout << "*********** afterIndex" << std::endl;
//      print(afterIndex, 120);
//      std::cout << afterIndex.sizes() << std::endl
//                << "strides: " << computeLinearStride(afterIndex)
//                << std::endl
//                << std::endl;
//
//      std::cout << "linearIndex += after" << std::endl;
//      print(linearIndex, 120);
//      std::cout << linearIndex.sizes() << std::endl
//                << "strides: " << computeLinearStride(linearIndex)
//                << std::endl
//                << std::endl;
//    std::cout << "sortedLinearIndex" << std::endl;
//                        print(sortedLinearIndex, 120);
//    std::cout << sortedLinearIndex.sizes() << std::endl
//                                  << "strides: " << computeLinearStride(sortedLinearIndex)
//                                  << std::endl
//                                  << std::endl;
