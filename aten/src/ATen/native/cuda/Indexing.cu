#include "hip/hip_runtime.h"
// Indexing tensors by by tensors
//
// This corresponds to "advanced indexing" in NumPy. The two operations are:
//
//  index(Tensor self, indices) -> Tensor
//  index_put_(Tensor self, indices, value, accumulate=false)
//
// The index is a TensorList containg kLong or kByte tensors or nulls. Byte
// tensors (boolean masks) are expanded to long tensors via nonzero(). Null
// tensors signify that the dimension is not indexed.
//
// All indexes are broadcast together and iterated as *one*. From NumPy:
//
// result[i_1, ..., i_M] == x[ind_1[i_1, ..., i_M], ind_2[i_1, ..., i_M],
//                           ..., ind_N[i_1, ..., i_M]]
//
// Note 1: ByteTensors expand to index as many dimensions as there are in the
// mask.
//
// Note 2: The behavior is more complicated when the index tensors are not all
// adjacent (e.g. x[[0, 1], :, [2, 3]]). In this case, self and the index
// tensors are transposed to the front: x.transpose(1, 2)[[0, 1], [2, 3]]
//
// The code contains two implementations of indexing. The more efficient
// implementation treats indexing like an elementwise operation over the
// tensors `result`, `x`, `ind_1`, `ind_2`, etc. This implementation does
// not work for index_put_ with accumulate=True. The other implementation
// combines the indexed tensors into a single linear index that is used
// with Tensor.put_. This is used for index_put_ with accumulate=True.
//
// The more efficient implementation takes the following steps for the
// above operation:
//
// 1) Broadcast ind_1, ind_2, ind_3 together to a common shape
// 2) Record x.stride(i) for each indexed dimension `i`
// 3) Replace the indexed subspace of `x` with the shape of the corresponding
//    subspace of `result` but with stride 0
// 4) Add dimensions of size 1 to the index tensors (ind_1, ind_2, etc.) so
//    that their shape is compatible with the result shape
//
// The CPU or CUDA kernel then computes element-wise over the broadcasted
// and restrided result, x, ind_1,  ind_2, etc.:
//
//   result[...] = *(&x[...] +
//                   ind_1[...] * x.stride(1) +
//                   ind_2[...] * x.stride(2) +
//                   ...)
//
// where & and * represent the C-style address-of and indirection operations.

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/ExpandUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Indexing.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <c10/core/ScalarType.h>

#include <ATen/native/Indexing.h>
#include <ATen/NativeFunctions.h>
#include <ATen/LegacyTHFunctions.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <hip/hip_vector_types.h>

#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <torch/csrc/utils/tensor_flatten.h>

#include <ATen/cpu/vec256/vec256.h>
#include <>

#ifdef __HIP_PLATFORM_HCC__
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif
#define GRID_SIZE 128
#define GROUP_SIZE 32

namespace at {
namespace native {

// FIXME
// MOVE THIS STUFF TO SOME COMMON PLACE SHARED BY .cpp and .cu

// STUFF BEGIN
[[noreturn]]
static void invalid_mask(const Tensor& self, int64_t idx, const Tensor& mask, int64_t maskIdx) {
  std::stringstream ss;
  ss << "The shape of the mask " << mask.sizes() << " at index " << maskIdx;
  ss << " does not match the shape of the indexed tensor " << self.sizes();
  ss << " at index " << idx;
  AT_INDEX_ERROR(ss.str());
}

static void checkIndexTensorTypes(TensorList indices) {
  for (auto& tensor : indices) {
    if (tensor.defined()) {
      auto scalarType = tensor.scalar_type();
      if (scalarType != kLong && scalarType != kByte) {
        AT_INDEX_ERROR("tensors used as indices must be long or byte tensors");
      }
    }
  }
}

static std::vector<Tensor> expandByteTensors(const Tensor& self, TensorList indices) {
  // Expands byte tensors (masks) into the equivalent indexing by LongTensors
  std::vector<Tensor> result;
  for (auto& index : indices) {
    if (index.scalar_type() == kByte) {
      // The sizes of the ByteTensor mask must match the sizes of the
      // corresponding dimensions in self
      for (int64_t j = 0; j < index.dim(); j++) {
        int64_t srcIdx = result.size() + j;
        if (index.size(j) != self.size(srcIdx)) {
          invalid_mask(self, srcIdx, index, j);
        }
      }
      // Replace with nonzeros
      auto nonzero = index.nonzero();
      auto special_empty = false;
      for (int64_t j = 0; j < index.dim(); j++) {
        if (special_empty) {
          // We can't call select on an empty tensor so we just create an empty
          // tensor.
          result.emplace_back(at::empty({0}, nonzero.options()));
        } else {
          result.emplace_back(nonzero.select(1, j));
        }
      }
    } else {
      result.emplace_back(index);
    }
  }
  return result;
}

static bool hasContiguousSubspace(TensorList tl) {
  // true if all the non-null tensors are adjacent
  auto isDefined = [](const Tensor& tensor) { return tensor.defined(); };
  auto isNull = [](const Tensor& tensor) { return !tensor.defined(); };
  auto start = std::find_if(tl.begin(), tl.end(), isDefined);
  auto stop = std::find_if(tl.rbegin(), tl.rend(), isDefined);
  auto it = std::find_if(start, stop.base(), isNull);
  return it == stop.base();
}

// Transposes the tensor and indices together so that all the non-null indices
// index the first k dimensions of the tensor. Returns the transposed tensor
// and the reordered indices. For example:
//  transposeToFront(tensor, {nullptr, a, nullptr, b})
// returns
//  tensor.permute([1, 3, 0, 2]), {a, b, nullptr, nullptr}
static std::tuple<Tensor, std::vector<Tensor>> transposeToFront(Tensor self, TensorList indices) {
  std::vector<int64_t> dims;
  std::vector<Tensor> transposedIndices;
  dims.reserve(self.dim());
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back(indices[i]);
    }
  }
  for (int64_t i = 0; i < self.dim(); i++) {
    if (!indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back();
    }
  }
  return std::make_tuple(self.permute(dims), std::move(transposedIndices));
}

static std::vector<int64_t> computeLinearStride(const Tensor& tensor) {
  // computes the stride as if tensor were contigous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1,
      std::multiplies<int64_t>());
  return stride;
}

// Unsqueezes src `before` times at the front and `after` times at the end
static Tensor unsqueezeN(const Tensor& src, int64_t before, int64_t after) {
  auto srcSizes = src.sizes();
  auto nDim = src.dim();
  std::vector<int64_t> sizes(nDim + before + after, 1);
  for (int64_t i = 0; i < nDim; i++) {
    sizes[i + before] = srcSizes[i];
  }
  return src.view(sizes);
}

// STUFF END


__global__ void arange_kernel(int64_t n, int64_t* a) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
      i += blockDim.x * gridDim.x) {
    a[i] = i;
  }
}

static std::tuple<Tensor, Tensor, Tensor, Tensor, int64_t, int64_t, int64_t, int64_t>
makeLinearIndex(Tensor self, TensorList orig) {
  checkIndexTensorTypes(orig);
  // first expand ByteTensor (boolean masks) into 1 or more LongTensors
  auto indices = expandByteTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t) self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices) = transposeToFront(self, indices);
  }

  auto strides = computeLinearStride(self);

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1;
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      Tensor index = (indices[i].remainder(self.size(i)) * strides[i]).to(kLong);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= self.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= self.size(i);
    }
  }

  // Compute the linear indices for the parts of the tensor not being indexed
  // ...and not being sorted
  Tensor beforeIndex;
  if (emptyBefore > 0) {
    Tensor index = at::native::empty_cuda({nElemBefore},
        self.options().dtype(kLong).device(at::DeviceType::CUDA));
    int64_t *pData = index.data<int64_t>();
    void* args[] = {&nElemBefore, &pData};
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    THCudaCheck(
        hipLaunchKernel((const void*) &arange_kernel,
            WARP_SIZE, GRID_SIZE, args, 0, stream));
    THCudaCheck(hipStreamSynchronize(stream));
    index = index * strides[emptyBefore - 1];
    index = index.view(self.sizes().slice(0, emptyBefore));
    beforeIndex = unsqueezeN(index, 0, linearIndex.dim() + emptyAfter);
  }
  Tensor afterIndex;
  if (emptyAfter > 0) {
    Tensor index = at::native::empty_cuda({nElemAfter},
        self.options().dtype(kLong).device(at::DeviceType::CUDA));
    int64_t *pData = index.data<int64_t>();
    void* args[] = {&nElemAfter, &pData};
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    THCudaCheck(
        hipLaunchKernel((const void*) &arange_kernel,
            WARP_SIZE, GRID_SIZE, args, 0, stream));
    THCudaCheck(hipStreamSynchronize(stream));
    index = index.view(self.sizes().slice(self.dim() - emptyAfter, emptyAfter));
    afterIndex = unsqueezeN(index, linearIndex.dim() + emptyBefore, 0);
  }

  return std::make_tuple(self, linearIndex, beforeIndex, afterIndex, emptyBefore, emptyAfter,
      nElemBefore, nElemAfter);
}

template<typename T, typename IndexType>
__device__ __forceinline__ IndexType
indexToOffset(const at::cuda::detail::TensorInfo<T, IndexType>& info, IndexType linearIndex) {
  IndexType offset(0);
  for (int i = info.dims - 1; i > 0; --i) {
    offset += (linearIndex % info.sizes[i]) * info.strides[i];
    linearIndex /= info.sizes[i];
  }
  return offset + linearIndex * info.strides[0];
}

//template<typename IndexType>
//__device__ __forceinline__
//IndexType indexToOffset(IndexType dims, IndexType* sizes, IndexType* strides,
//    IndexType linearIndex) {
//  IndexType offset(0);
//  for (IndexType i = dims - 1; i > 0; --i) {
//    offset += (linearIndex % sizes[i]) * strides[i];
//    linearIndex /= sizes[i];
//  }
//  return offset + linearIndex* strides[0];
//}

template<typename index_t>
__device__ __forceinline__
index_t extended_pos(index_t nseq, index_t baStride, index_t sortedSize,
    const int64_t* origOrder) {
  return nseq / sortedSize + origOrder[nseq % sortedSize] * baStride;
}

template<typename index_t>
__device__ __forceinline__
index_t extended_pos(index_t idx, index_t blockSize, index_t currentThreadInBlock,
    index_t baStride, index_t sortedSize, const int64_t* origOrder) {
  const index_t nseq = idx * blockSize + currentThreadInBlock;
  return extended_pos(nseq, baStride, sortedSize, origOrder);
}

template<typename scalar_t>
__global__
void backward_indexing_kernel(const int64_t* extendedIdx,
    int64_t* origOrder, scalar_t* gradValues, int64_t extendedIdxSize,
    int64_t baStride, int64_t sortedSize, scalar_t* dstData) {
  using accscalar_t = acc_type<scalar_t, true>;

  int blockSize = blockDim.x * blockDim.y * blockDim.z;
  int idxMax = (extendedIdxSize + blockSize - 1) / blockSize;
  int idx = blockIdx.x;

  int blockHeadPos = extended_pos<int64_t>(idx, blockSize, 0L,
      baStride, sortedSize, origOrder);

//  printf("BBBBBBBB blockHeadPos %d \n", blockHeadPos); // TODO!!!

  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < extendedIdxSize;
       i += blockDim.x * gridDim.x) {

    if (blockHeadPos < extendedIdxSize &&
        (idx == 0 || blockHeadPos != extended_pos<int64_t>(idx - 1, blockSize, 0L,
                                                           baStride, sortedSize, origOrder))) {
      int th = i % GROUP_SIZE;
      const int no = extended_pos<int>(
          idx, blockSize, th, baStride, sortedSize, origOrder);

      //       printf("****** %d %d %d %d\n", idx, i, th, no); // TODO!!!

      __shared__ int offsetArr[GROUP_SIZE];
      __shared__ accscalar_t valArr[GROUP_SIZE];

      offsetArr[th] = extendedIdx[no];
      valArr[th] = gradValues[no];
      __syncthreads();
      if (th == 0 && idx < idxMax) {
        int currentBlockEnd = extendedIdxSize - idx * GROUP_SIZE;
        currentBlockEnd =
            currentBlockEnd < GROUP_SIZE ? currentBlockEnd : GROUP_SIZE;

        //     printf("                %d %d %d %d\n", idx, i, th, currentBlockEnd); // TODO!!!

        for (int t = 0; t < currentBlockEnd; ++t) {
          dstData[offsetArr[t]] += valArr[t];

          //        printf("%d %d %d\n", idx, t, offsetArr[t]);
        }
        __threadfence();
      }
      ++idx;
      const int blockHeadPosNext = extended_pos<int64_t>(
          idx, blockSize, 0L, baStride, sortedSize, origOrder);
      if (blockHeadPosNext >= extendedIdxSize ||
          blockHeadPosNext != blockHeadPos) {
        // next block is processing other images' indexes, exit.
        break;
      }
      blockHeadPos = blockHeadPosNext; // keep going in the same warp
    }
  }
//  __threadfence_block();
}


/*

  //int idx = blockIdx.x * GROUP_SIZE + threadIdx.y;
  int blockSize = blockDim.x * blockDim.y * blockDim.z;

//  if (idx % GROUP_SIZE > 0) return;

  int blockHeadPos = (int) extended_pos<int64_t>(idx, baStride, sortedSize, origOrder);


  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < extendedIdxSize; i += blockDim.x * gridDim.x) {
    int th = i % WARP_SIZE;



  if (i < extendedIdxSize && (i == 0 || blockHeadPos !=
      extended_pos<int64_t>(i - 1, baStride, sortedSize, origOrder))) {


    do {
//      const int ft = threadIdx.x + blockIdx.y * blockDim.x * GROUP_SIZE;


//      int offsetArr[GROUP_SIZE];
//      accscalar_t valArr[GROUP_SIZE];

      #pragma unroll
      for (int g = 0; g < GROUP_SIZE; g++) {

//        int feature_dim = ft + i * WARP_SIZE;
//        const int th = i + i;

        printf("****** %d %d %d   %d %d \n", idx, i, th,
               ft, feature_dim); // TODO!!!

        const int no = extended_pos<int>(i + g, baStride, sortedSize, origOrder);
//        offsetArr[i] = extendedIdx[no];
//        valArr[i] = gradValues[no];

        dstData[extendedIdx[no]] += gradValues[no];

      }
      idx++;
      const int blockHeadPosNext =
          extended_pos<int64_t>(idx, baStride, sortedSize, origOrder);
      if (blockHeadPosNext >= extendedIdxSize || blockHeadPosNext != blockHeadPos) {
        // next block is processing other images' indexes, exit.
        break;
      }
      blockHeadPos = blockHeadPosNext;  // keep going in the same warp
    } while (true);
  }
*/

template<typename T>
struct TensorAccumFullyIndexedPutOp : thrust::unary_function<int64_t, T> {
  TensorAccumFullyIndexedPutOp(at::cuda::detail::TensorInfo<T, int64_t> info, const T* psrc,
      const int64_t* sorted_idx_beg, const int64_t* sorted_idx_end, const int64_t* orig_idx_beg)
      : info(info), p_source(psrc), sorted_idx_beg(sorted_idx_beg), sorted_idx_end(sorted_idx_end),
        orig_idx_beg(orig_idx_beg) {}

  __device__ __forceinline__ T

  operator()(int64_t n) {
    const int64_t no = orig_idx_beg[n]; // restore if flipped
    const T* pvalue = p_source + no;
    const int64_t* pindex = sorted_idx_beg + no;
    const int64_t linear_index = *pindex;
    const int64_t offset = indexToOffset<T, int64_t>(info, linear_index);

    if (pindex == sorted_idx_beg || *pindex != *(pindex - 1)) {
      do {
        info.data[offset] += *pvalue;
        pindex++;
        pvalue++;
      } while (pindex != sorted_idx_end && *pindex == linear_index);
    }
    return 0; // discarded
  }

  const at::cuda::detail::TensorInfo<T, int64_t> info;
  const T* p_source;
  const int64_t* sorted_idx_beg;
  const int64_t* sorted_idx_end;
  const int64_t* orig_idx_beg;
};

Tensor& index_put_cuda_(Tensor& self_, TensorList indices, const Tensor& value, bool accumulate) {
  if (indices.size() > (size_t) self_.dim()) {
    AT_INDEX_ERROR("too many indices for tensor of dimension ", self_.dim(), " (got ",
        indices.size(), ")");
  }
  Tensor self, linearIndex;
  Tensor beforeIndex, afterIndex;
  int64_t emptyBefore = 0L, emptyAfter = 0L;
  int64_t nElemBefore = 1L, nElemAfter = 1L;

  std::tie(self, linearIndex, beforeIndex, afterIndex, emptyBefore, emptyAfter, nElemBefore,
      nElemAfter) = makeLinearIndex(self_, indices);

  auto sortedLinearIndex = linearIndex.clone();
  auto origCounters = at::empty_like(linearIndex);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  int64_t idxSize = linearIndex.numel(); // const breaks usin cudaKernelLaunch
  int64_t dstSize = self.numel();
  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "xput_cuda_1", [&] {
    int64_t* sortedLinearIndex_beg = sortedLinearIndex.data<int64_t>();
    int64_t* sortedLinearIndex_end = sortedLinearIndex_beg + idxSize;
    int64_t* origCounters_beg = origCounters.data<int64_t>();
    auto sortedLinearIndex_iter = thrust::device_ptr<int64_t>(sortedLinearIndex_beg);
    auto origCounters_iter = thrust::device_ptr<int64_t>(origCounters_beg);
    auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
    TensorAccumFullyIndexedPutOp<scalar_t> aiPutOp(self_info, value.data<scalar_t>(),
        sortedLinearIndex_beg, sortedLinearIndex_end, origCounters_beg);

    thrust::sequence(policy, origCounters_iter, origCounters_iter + idxSize);

    thrust::sort_by_key(policy, sortedLinearIndex_iter, sortedLinearIndex_iter + idxSize,
        origCounters_iter, ThrustLTOp<int64_t>());

    if (!beforeIndex.defined() && !afterIndex.defined()) {
      // Full size index, done:
      thrust::counting_iterator<int64_t> first(0);
      thrust::counting_iterator<int64_t> last(idxSize);
      thrust::for_each(policy, first, last, aiPutOp);
    }
  });

  if (beforeIndex.defined() || afterIndex.defined()) {
    // Sum with broadcasting to compute the full index
    // using unsorted original
    linearIndex = unsqueezeN(linearIndex, emptyBefore, emptyAfter);
    if (emptyBefore > 0) {
      linearIndex = linearIndex + beforeIndex;
    }
    if (emptyAfter > 0) {
      linearIndex = linearIndex + afterIndex;
    }

//    linearIndex = linearIndex.flatten();
//
//
//    std::cout << "linearIndex += before" << std::endl;
//print(linearIndex, 120);
//std::cout << linearIndex.sizes() << std::endl
//<< "strides: " << computeLinearStride(linearIndex)
//<< std::endl
//<< std::endl;



    AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "index_put_cuda_kernel_", [&] {
      cuda::detail::TensorInfo <scalar_t, int64_t> self_info =
          cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
      scalar_t* valuePtr = value.data<scalar_t>();
      int64_t extendedIdxSize = linearIndex.numel();
      int64_t* origCountersPtr = origCounters.data<int64_t>();
      int64_t* extendedLinearIndexPtr = linearIndex.data<int64_t>();
      int64_t baStride = nElemAfter * nElemBefore;

         //    printf("############# %lld \n", baStride); // TODO!!!

//      int blockSize = WARP_SIZE;
      dim3 blockSize(GROUP_SIZE); //, GROUP_SIZE);WARP_SIZE,
      dim3 gridSize(GRID_SIZE); //(extendedIdxSize + GROUP_SIZE - 1) / GROUP_SIZE);
//      dim3 gridSize((extendedIdxSize + GROUP_SIZE - 1) / GROUP_SIZE);
  //        THCCeilDiv(num_indices, (int64_t) 4), THCCeilDiv(stride, (int64_t) 128));

      //int gridSize = GRID_SIZE;// (extendedIdxSize + blockSize - 1) / blockSize;
      void* args[] = {&extendedLinearIndexPtr, &origCountersPtr, &valuePtr, &extendedIdxSize,
                      &baStride, &idxSize, &self_info.data};
      THCudaCheck(
          hipLaunchKernel((const void*) &backward_indexing_kernel<scalar_t>, gridSize, blockSize,
              args, 0, stream));
      THCudaCheck(hipStreamSynchronize(stream));
      THCudaCheck(hipGetLastError());
    });
  }
  return self_;
}

}}






//long cnt = 0L;
//long total = 0L;
//
//

//++cnt;
//auto start = std::chrono::high_resolution_clock::now();
//
//auto finish = std::chrono::high_resolution_clock::now();
//total += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//if (cnt % 100 == 0) {
//std::cout << "AFTER indices: " << total / cnt << "ns" << std::endl;
//}

//    std::cout << "afterIndex" << std::endl;
//                        print(afterIndex, 120);
//                        std::cout << std::endl
//                                  << "strides: " << computeLinearStride(afterIndex)
//                                  << std::endl
//                                  << std::endl;

//std::cout << "*********** beforeIndex" << std::endl;
//print(beforeIndex, 120);
//std::cout << beforeIndex.sizes() << std::endl
//<< "strides: " << computeLinearStride(beforeIndex)
//<< std::endl
//<< std::endl;
//
//std::cout << "linearIndex += before" << std::endl;
//print(linearIndex, 120);
//std::cout << linearIndex.sizes() << std::endl
//<< "strides: " << computeLinearStride(linearIndex)
//<< std::endl
//<< std::endl;

//std::cout << "self" << std::endl;
//print(self, 120);
//std::cout << self.sizes() << std::endl
//<< "strides: " << computeLinearStride(self)
//<< std::endl
//<< std::endl;
//
//std::cout << "origCounters" << std::endl;
//print(origCounters, 120);
//std::cout << origCounters.sizes() << std::endl
//<< "strides: " << computeLinearStride(origCounters)
//<< std::endl
//<< std::endl;

//      std::cout << "*********** afterIndex" << std::endl;
//      print(afterIndex, 120);
//      std::cout << afterIndex.sizes() << std::endl
//                << "strides: " << computeLinearStride(afterIndex)
//                << std::endl
//                << std::endl;
//
//      std::cout << "linearIndex += after" << std::endl;
//      print(linearIndex, 120);
//      std::cout << linearIndex.sizes() << std::endl
//                << "strides: " << computeLinearStride(linearIndex)
//                << std::endl
//                << std::endl;
//    std::cout << "sortedLinearIndex" << std::endl;
//                        print(sortedLinearIndex, 120);
//    std::cout << sortedLinearIndex.sizes() << std::endl
//                                  << "strides: " << computeLinearStride(sortedLinearIndex)
//                                  << std::endl
//                                  << std::endl;
