#include "hip/hip_runtime.h"
// Indexing tensors by by tensors
//
// This corresponds to "advanced indexing" in NumPy. The two operations are:
//
//  index(Tensor self, indices) -> Tensor
//  index_put_(Tensor self, indices, value, accumulate=false)
//
// The index is a TensorList containg kLong or kByte tensors or nulls. Byte
// tensors (boolean masks) are expanded to long tensors via nonzero(). Null
// tensors signify that the dimension is not indexed.
//
// All indexes are broadcast together and iterated as *one*. From NumPy:
//
// result[i_1, ..., i_M] == x[ind_1[i_1, ..., i_M], ind_2[i_1, ..., i_M],
//                           ..., ind_N[i_1, ..., i_M]]
//
// Note 1: ByteTensors expand to index as many dimensions as there are in the
// mask.
//
// Note 2: The behavior is more complicated when the index tensors are not all
// adjacent (e.g. x[[0, 1], :, [2, 3]]). In this case, self and the index
// tensors are transposed to the front: x.transpose(1, 2)[[0, 1], [2, 3]]
//
// The code contains two implementations of indexing. The more efficient
// implementation treats indexing like an elementwise operation over the
// tensors `result`, `x`, `ind_1`, `ind_2`, etc. This implementation does
// not work for index_put_ with accumulate=True. The other implementation
// combines the indexed tensors into a single linear index that is used
// with Tensor.put_. This is used for index_put_ with accumulate=True.
//
// The more efficient implementation takes the following steps for the
// above operation:
//
// 1) Broadcast ind_1, ind_2, ind_3 together to a common shape
// 2) Record x.stride(i) for each indexed dimension `i`
// 3) Replace the indexed subspace of `x` with the shape of the corresponding
//    subspace of `result` but with stride 0
// 4) Add dimensions of size 1 to the index tensors (ind_1, ind_2, etc.) so
//    that their shape is compatible with the result shape
//
// The CPU or CUDA kernel then computes element-wise over the broadcasted
// and restrided result, x, ind_1,  ind_2, etc.:
//
//   result[...] = *(&x[...] +
//                   ind_1[...] * x.stride(1) +
//                   ind_2[...] * x.stride(2) +
//                   ...)
//
// where & and * represent the C-style address-of and indirection operations.

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/ExpandUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <c10/core/ScalarType.h>

#include <ATen/native/Indexing.h>
#include <ATen/NativeFunctions.h>
#include <ATen/LegacyTHFunctions.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <torch/csrc/utils/tensor_flatten.h>

#include <ATen/cpu/vec256/vec256.h>
#include "../../../../../c10/core/DeviceType.h"

namespace at { namespace native {

[[noreturn]]
static void invalid_mask(const Tensor & self, int64_t idx, const Tensor & mask, int64_t maskIdx) {
  std::stringstream ss;
  ss << "The shape of the mask " << mask.sizes() << " at index " << maskIdx;
  ss << " does not match the shape of the indexed tensor " << self.sizes();
  ss << " at index " << idx;
  AT_INDEX_ERROR(ss.str());
}

static void checkIndexTensorTypes(TensorList indices) {
  for (auto& tensor : indices) {
    if (tensor.defined()) {
      auto scalarType = tensor.scalar_type();
      if (scalarType != kLong && scalarType != kByte) {
        AT_INDEX_ERROR("tensors used as indices must be long or byte tensors");
      }
    }
  }
}

static std::vector<Tensor> expandByteTensors(const Tensor & self, TensorList indices) {
  // Expands byte tensors (masks) into the equivalent indexing by LongTensors
  std::vector<Tensor> result;
  for (auto & index : indices) {
    if (index.scalar_type() == kByte) {
      // The sizes of the ByteTensor mask must match the sizes of the
      // corresponding dimensions in self
      for (int64_t j = 0; j < index.dim(); j++) {
        int64_t srcIdx = result.size() + j;
        if (index.size(j) != self.size(srcIdx)) {
          invalid_mask(self, srcIdx, index, j);
        }
      }
      // Replace with nonzeros
      auto nonzero = index.nonzero();
      auto special_empty = false;
      for (int64_t j = 0; j < index.dim(); j++) {
        if (special_empty) {
          // We can't call select on an empty tensor so we just create an empty
          // tensor.
          result.emplace_back(at::empty({0}, nonzero.options()));
        } else {
          result.emplace_back(nonzero.select(1, j));
        }
      }
    } else {
      result.emplace_back(index);
    }
  }
  return result;
}

static bool hasContiguousSubspace(TensorList tl) {
  // true if all the non-null tensors are adjacent
  auto isDefined = [](const Tensor & tensor){ return tensor.defined(); };
  auto isNull = [](const Tensor & tensor){ return !tensor.defined(); };
  auto start = std::find_if(tl.begin(), tl.end(), isDefined);
  auto stop = std::find_if(tl.rbegin(), tl.rend(), isDefined);
  auto it = std::find_if(start, stop.base(), isNull);
  return it == stop.base();
}

// Transposes the tensor and indices together so that all the non-null indices
// index the first k dimensions of the tensor. Returns the transposed tensor
// and the reordered indices. For example:
//  transposeToFront(tensor, {nullptr, a, nullptr, b})
// returns
//  tensor.permute([1, 3, 0, 2]), {a, b, nullptr, nullptr}
static std::tuple<Tensor, std::vector<Tensor>>
transposeToFront(Tensor self, TensorList indices) {
  std::vector<int64_t> dims;
  std::vector<Tensor> transposedIndices;
  dims.reserve(self.dim());
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back(indices[i]);
    }
  }
  for (int64_t i = 0; i < self.dim(); i++) {
    if (!indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back();
    }
  }
  return std::make_tuple(self.permute(dims), std::move(transposedIndices));
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contigous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

// Unsqueezes src `before` times at the front and `after` times at the end
static Tensor unsqueezeN(const Tensor & src, int64_t before, int64_t after) {
  auto srcSizes = src.sizes();
  auto nDim = src.dim();
  std::vector<int64_t> sizes(nDim + before + after, 1);
  for (int64_t i = 0; i < nDim; i++) {
    sizes[i + before] = srcSizes[i];
  }
  return src.view(sizes);
}

static std::tuple<Tensor, Tensor, Tensor, Tensor, int64_t, int64_t, int64_t, int64_t>
    makeLinearIndex(Tensor self, TensorList orig) {
  checkIndexTensorTypes(orig);
  // first expand ByteTensor (boolean masks) into 1 or more LongTensors
  auto indices = expandByteTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices) = transposeToFront(self, indices);
  }

  auto strides = computeLinearStride(self);

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1;
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      if (linearIndex.defined()) {
        linearIndex += indices[i].remainder(self.size(i)) * strides[i];
      } else {
        linearIndex = indices[i].remainder(self.size(i)) * strides[i];
      }
    }
    else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= self.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= self.size(i);
    }
  }
  // Compute the linear indices for the parts of the tensor not being indexed
  // ...and not being sorted
  Tensor beforeIndex;
  if (emptyBefore > 0) {
    beforeIndex = at::arange(0, nElemBefore, self.options().dtype(kLong)) * strides[emptyBefore - 1];
    beforeIndex = beforeIndex.view(self.sizes().slice(0, emptyBefore));
    beforeIndex = unsqueezeN(beforeIndex, 0, linearIndex.dim() + emptyAfter);
  }
  Tensor afterIndex;
  if (emptyAfter > 0) {
    afterIndex = at::arange(0, nElemAfter, self.options().dtype(kLong));
    afterIndex = afterIndex.view(self.sizes().slice(self.dim() - emptyAfter, emptyAfter));
    afterIndex = unsqueezeN(afterIndex, linearIndex.dim() + emptyBefore, 0);
  }
  return std::make_tuple(self, linearIndex, beforeIndex, afterIndex,
                         emptyBefore, emptyAfter, nElemBefore, nElemAfter);
}

template <typename T, typename IndexType>
__device__ __forceinline__ IndexType indexToOffset(
    const at::cuda::detail::TensorInfo<T, IndexType>& info, IndexType linearIndex) {
  IndexType offset(0);
  for (int i = info.dims - 1; i > 0; --i) {
    offset += (linearIndex % info.sizes[i]) * info.strides[i];
    linearIndex /= info.sizes[i];
  }
  return offset + linearIndex * info.strides[0];
}

template <typename T>
struct TensorAccumMixedPutOp : thrust::unary_function<int64_t, T> {
  TensorAccumMixedPutOp(at::cuda::detail::TensorInfo<T, int64_t> info,
  const T* psrc,
  const int64_t* ext_idx_beg, const int64_t* ext_idx_end,
  const int64_t* orig_idx_beg, int64_t sortedStride,
  int64_t sortedSize)
  : info(info), p_source(psrc),
  ext_idx_beg(ext_idx_beg), ext_idx_end(ext_idx_end),
  orig_idx_beg(orig_idx_beg), sortedStride(sortedStride),
  sortedSize(sortedSize) {}

//  __device__ __forceinline__
//  T operator()(thrust::tuple <int64_t, T> tuple_idx) {
//    const int64_t idx = thrust::get<0>(tuple_idx);
//    const int64_t offset = indexToOffset < T, int64_t>(info, idx);
//    const T value = thrust::get<1>(tuple_idx);  // orig_idx
//    info.data[offset] += value;
//    return 0;  // discarded
//  }

//  int64_t no = orig_idx_beg[n / lindexStride] * lindexStride + (n % lindexStride);

  __device__ __forceinline__
  T operator()(int64_t n) {

    printf("%lld %lld %lld %lld \n",n , sortedStride, n / sortedStride, orig_idx_beg[n]);

    const int64_t no = orig_idx_beg[n / sortedStride] * sortedStride + (n % sortedStride);
//    const int64_t ino = no / sortedSize;
    const T* pvalue = p_source + no;
    const int64_t* pindex = ext_idx_beg + no;
    const int64_t linear_index = *pindex;
    const int64_t offset = indexToOffset<T, int64_t>(info, linear_index);

    printf("%lld %lld %lld %lld %g\n", n , no, linear_index, offset, *pvalue);


    if (pindex == ext_idx_beg || *pindex != *(pindex - 1)) {
      do {
        info.data[offset] += *pvalue;
        pindex++;
        pvalue++;

        printf("...%lld %lld %lld %g\n", n , no, *pindex, offset, info.data[offset]);
      } while (pindex != ext_idx_end && *pindex == linear_index);
    }
    return 0;  // discarded
  }

  at::cuda::detail::TensorInfo<T, int64_t> info;
  const T* p_source;
  const int64_t* ext_idx_beg;
  const int64_t* ext_idx_end;
  const int64_t* orig_idx_beg;
  const int64_t sortedStride;
  const int64_t sortedSize;
};

template <typename T>
struct TensorAccumFullyIndexedPutOp : thrust::unary_function<int64_t, T> {
  TensorAccumFullyIndexedPutOp(at::cuda::detail::TensorInfo<T, int64_t> info,
      const T* psrc,
      const int64_t* sorted_idx_beg, const int64_t* sorted_idx_end,
      const int64_t* orig_idx_beg)
    : info(info), p_source(psrc),
      sorted_idx_beg(sorted_idx_beg), sorted_idx_end(sorted_idx_end),
      orig_idx_beg(orig_idx_beg) {}

  __device__ __forceinline__ T operator()(int64_t n) {
    const int64_t no = orig_idx_beg[n];  // restore if flipped
    const T* pvalue = p_source + no;
    const int64_t* pindex = sorted_idx_beg + no;
    const int64_t linear_index = *pindex;
    const int64_t offset = indexToOffset<T, int64_t>(info, linear_index);

    if (pindex == sorted_idx_beg || *pindex != *(pindex - 1)) {
      do {
        info.data[offset] += *pvalue;
        pindex++;
        pvalue++;
      } while (pindex != sorted_idx_end && *pindex == linear_index);
    }
    return 0;  // discarded
  }

  const at::cuda::detail::TensorInfo<T, int64_t> info;
  const T* p_source;
  const int64_t* sorted_idx_beg;
  const int64_t* sorted_idx_end;
  const int64_t* orig_idx_beg;
};

Tensor & index_put_cuda_(Tensor & self_, TensorList indices, const Tensor & value, bool accumulate) {
  if (indices.size() > (size_t)self_.dim()) {
    AT_INDEX_ERROR("too many indices for tensor of dimension ", self_.dim(), " (got ", indices.size(), ")");
  }

  Tensor self, linearIndex, expandedValue;
  Tensor beforeIndex, afterIndex;
  int64_t emptyBefore = 0L, emptyAfter = 0L;
  int64_t nElemBefore = 1L, nElemAfter = 1L;

  std::tie(self, linearIndex, beforeIndex, afterIndex,
      emptyBefore, emptyAfter, nElemBefore, nElemAfter) = makeLinearIndex(self_, indices);

  auto sortedLinearIndex = linearIndex;
  auto origCounters = at::empty_like(linearIndex);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  const int64_t idxSize = linearIndex.numel();
  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "xput_cuda_1", [&] {
    int64_t* sortedLinearIndex_beg = sortedLinearIndex.data<int64_t>();
    int64_t* sortedLinearIndex_end = sortedLinearIndex_beg + idxSize;
    int64_t* origCounters_beg = origCounters.data<int64_t>();
    auto sortedLinearIndex_iter = thrust::device_ptr<int64_t>(sortedLinearIndex_beg);
    auto origCounters_iter = thrust::device_ptr<int64_t>(origCounters_beg);
    auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
    TensorAccumFullyIndexedPutOp<scalar_t> aiPutOp(self_info,
        value.data<scalar_t>(), sortedLinearIndex_beg, sortedLinearIndex_end,
        origCounters_beg);

    thrust::sequence(policy,
        origCounters_iter, origCounters_iter + idxSize);

    thrust::sort_by_key(
        policy,
        sortedLinearIndex_iter, sortedLinearIndex_iter + idxSize,
        origCounters_iter, ThrustLTOp<int64_t>());

    std::cout << "origCounters" << std::endl;
    print(origCounters, 120);
    std::cout << std::endl << "strides: "
              << computeLinearStride(origCounters) << std::endl << std::endl;
    std::cout << "value" << std::endl;
    print(value, 120);
    std::cout << std::endl << "strides: "
              << computeLinearStride(value) << std::endl << std::endl;
    std::cout << "sortedLinearIndex" << std::endl;
    print(sortedLinearIndex, 120);
    std::cout << std::endl << "strides: "
              << computeLinearStride(sortedLinearIndex) << std::endl << std::endl;

    if (!beforeIndex.defined() && !afterIndex.defined()) {
      // Full size index, done:
      thrust::counting_iterator<int64_t> first(0);
      thrust::counting_iterator<int64_t> last(idxSize);
      thrust::for_each(policy, first, last, aiPutOp);
    }
  });

  if (beforeIndex.defined() || afterIndex.defined()) {
    Tensor extendedLinearIndex;
    // Sum with broadcasting to compute the full index
    linearIndex = unsqueezeN(linearIndex, emptyBefore, emptyAfter);
    if (emptyBefore > 0) {
      beforeIndex = unsqueezeN(beforeIndex, 0, linearIndex.dim() + emptyAfter);

      std::cout << "beforeIndex" << std::endl;
      print(beforeIndex, 120);
      std::cout << std::endl << "strides: "
                << computeLinearStride(beforeIndex) << std::endl << std::endl;

      extendedLinearIndex = linearIndex + beforeIndex;
    }
    if (emptyAfter > 0) {
      assert(false);
      afterIndex = unsqueezeN(afterIndex, linearIndex.dim() + emptyBefore, 0);

      std::cout << "afterIndex" << std::endl;
      print(afterIndex, 120);
      std::cout << std::endl << "strides: "
                << computeLinearStride(afterIndex) << std::endl << std::endl;

      linearIndex = linearIndex + afterIndex;

    }
    extendedLinearIndex.squeeze_();

    std::cout << "extendedLinearIndex" << std::endl;
    print(extendedLinearIndex, 120);
    std::cout << std::endl << "strides: "
              << computeLinearStride(extendedLinearIndex) << std::endl << std::endl;

    AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "index_put_cuda_2", [&] {
      const scalar_t *pvalue = value.data<scalar_t>();
      const int64_t extendedIdxSize = extendedLinearIndex.numel();
      int64_t* extendedLinearIndex_beg = extendedLinearIndex.data<int64_t>();
      int64_t* extendedLinearIndex_end = extendedLinearIndex_beg + extendedIdxSize;
      int64_t* origCounters_beg = origCounters.data<int64_t>();
      auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
      TensorAccumMixedPutOp<scalar_t> amPutOp(self_info,
        pvalue, extendedLinearIndex_beg, extendedLinearIndex_end,
        origCounters_beg, extendedIdxSize / idxSize, idxSize);

//      auto first = thrust::make_zip_iterator(
//          thrust::make_tuple(plindex, pvalue));
//      auto last = thrust::make_zip_iterator(
//          thrust::make_tuple(plindex + extendedIdxSize, pvalue + extendedIdxSize));
      thrust::counting_iterator<int64_t> first(0);
      thrust::counting_iterator<int64_t> last(extendedIdxSize);
      thrust::for_each(policy, first, last, amPutOp);

    });
  }
  return self_;
}

}}




//  printf("%lld %lld %lld %lld %g\n", n , no, linear_index,
//  offset, *pvalue);
//  int64_t no = orig_idx_beg[n / lindexStride] * lindexStride + (n % lindexStride);
//  const int64_t lindex = plindex[no];
//  const int64_t headIdx = lindex / headStride;
//  const int64_t shPos = lindex % headStride;
//  const int64_t offset = headIdx * headStride + shPos;
//
//  printf("%lld %lld %lld %lld %lld %lld %lld\n", n , no, lindex, headStride, headIdx, shPos, offset);
//  int64_t sortedIndexSize;
//  const int64_t* beforeIndex;
//  const int64_t elementsBefore;
//  const int64_t* afterIndex;
//  const int64_t elementsAfter;


//  std::cout << "strides" << std::endl;
//  std::cout << strides << std::endl;
//  std::cout << std::endl << std::endl;

//    std::cout << "beforeIndex" << std::endl;
//    print(beforeIndex, 120);
//    std::cout << std::endl << "strides: "
//              << computeLinearStride(beforeIndex) << std::endl << std::endl;

//    std::cout << "afterIndex" << std::endl;
//    print(afterIndex, 120);
//    std::cout << std::endl << "strides: "
//              << computeLinearStride(afterIndex) << std::endl << std::endl;
//  std::cout << "linearIndex" << std::endl;
//  print(linearIndex, 120);
//  std::cout << std::endl << "strides: "
//      << computeLinearStride(linearIndex) << std::endl << std::endl;

//  Tensor lll = unsqueezeN(linearIndex, emptyBefore, emptyAfter) + afterIndex;
//  std::cout << "lll" << std::endl;
//  print(lll, 120);
//  std::cout << std::endl << "strides: "
//            << computeLinearStride(linearIndex) << std::endl << std::endl;





//    const T* psrc, const int64_t* plindex)
//    const int64_t* orig_idx_beg, int64_t origIdxSize,
//      int64_t headSize, int64_t headStride, int64_t lindexStride)
//   , p_source(psrc), plindex(plindex) {}
//      orig_idx_beg(orig_idx_beg), origIdxSize(origIdxSize),
//      headSize(headSize), headStride(headStride), lindexStride(lindexStride) {}


//    atomicAdd(address, value);
//    T* address = info.data + offset;
//    atomicAdd(address, value);

//__device__ __forceinline__ T operator()(int64_t n) {
//
//
//    const T value = p_source[n];
////    const int64_t lindex = plindex[n];
//    const int64_t offset = indexToOffset(info, plindex[n]);
//    T* address = info.data + offset;
//
////    printf("%lld %lld %lld %g %g\n", n , lindex, offset,
////        info.data[offset], value);
//
//    atomicAdd(address, value);
//
//  return 0;  // discarded
//  }


//  const T* p_source;
//  const int64_t* plindex;
//  const int64_t* orig_idx_beg;
//  const int64_t origIdxSize;
//  const int64_t headSize; // size of sorted linearIndex (aka "head")
//  const int64_t headStride; // distance between shoulder slices
//  const int64_t lindexStride; // full index major stride

//long long mtotal = 0;
//long long xtotal = 0;
//long pcnt = 0;

//////////////////////////////////////////////////
//  ++pcnt;
//  auto start = std::chrono::high_resolution_clock::now();
/////////////////////////////////////////////////////////////

//  auto finish = std::chrono::high_resolution_clock::now();
//
//  mtotal += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//  if (pcnt % 100 == 0) {
//    std::cout << "makeLinearIndex: "
//              << mtotal/pcnt/1000
//              << "us" << std::endl;;
//  }


//  std::tie(expandedValue) = expand_inplace(linearIndex, value);

//  start = std::chrono::high_resolution_clock::now();

//  Tensor & ret = src.xput_(linearIndex, value, accumulate, beforeIndex, afterIndex,
//                           emptyBefore, emptyAfter);
//  Tensor & xput_cuda_(Tensor & self, Tensor & linearIndex, const Tensor & source, bool accumulate,
//      Tensor & beforeIndex, Tensor & afterIndex, int64_t emptyBefore, int64_t emptyAfter)

//  std::cout << "value" << std::endl;
//  print(value, 120);
//  std::cout << std::endl << "strides: "
//            << computeLinearStride(value) << std::endl << std::endl;

//  std::cout << "self" << std::endl;
//  print(self, 120);
//  std::cout << std::endl << "strides: "
//            << computeLinearStride(self) << std::endl << std::endl;

//  auto sortedLinearIndex = linearIndex;//.clone();
//  auto orig_index = at::empty_like(linearIndex);


//  if (beforeIndex.defined() || afterIndex.defined()) {

//    const int64_t *beforeIndexPtr = beforeIndex.defined() ? beforeIndex.data<int64_t>() : nullptr;
//    const int64_t elementsBefore = beforeIndex.defined() ? beforeIndex.numel() : 0L;
//    const int64_t *afterIndexPtr = afterIndex.defined() ? afterIndex.data<int64_t>() : nullptr;
//    const int64_t elementsAfter = afterIndex.defined() ? afterIndex.numel() : 0L;


//      std::cout << "beforeIndex" << std::endl;
//      print(beforeIndex, 120);
//      std::cout << std::endl << "strides: "
//                << computeLinearStride(beforeIndex) << std::endl << std::endl;

//      std::cout << "afterIndex" << std::endl;
//      print(afterIndex, 120);
//      std::cout << std::endl << "strides: "
//                << computeLinearStride(afterIndex) << std::endl << std::endl;

//    std::cout << "linearIndex GPU" << std::endl;
//    print(linearIndex, 120);
//    std::cout << std::endl << "strides: "
//              << computeLinearStride(linearIndex) << std::endl << std::endl;


//    const int64_t* orig_idx_beg = orig_index.data<int64_t>();
//    const int64_t* orig_idx_end = orig_idx_beg + idxSize;
//    auto full_index_iter = thrust::device_ptr<int64_t>(linearIndex.data<int64_t>());
//    int64_t liStride = fullIdxSize / idxSize;
//    const int64_t headStride = dstSize / (nElemBefore * nElemAfter);

//, value.data<scalar_t>(), plindex);
//      , orig_idx_beg,
//          idxSize, idxSize, headStride, liStride);

//
//
//
//
//  finish = std::chrono::high_resolution_clock::now();
//
//  xtotal += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//  if (pcnt % 100 == 0) {
//    std::cout << "xput: "
//              << xtotal/pcnt/1000
//              << "us" << std::endl;;
//  }



//  return src.xput_(linearIndex, value, accumulate);

//      thrust::counting_iterator<int64_t> full_index_iter(0);
//      thrust::for_each_n(policy, full_index_iter, fullIdxSize, amPutOp);

//    printf("< %lld %lld info.data[lindex]=%g value=%g\n",
//        n, lindex, info.data[lindex], value);

//    if (elementsBefore == 0 && elementsAfter == 0) {
// all dimensions indexed
//      int64_t* pindex = &sorted_idx;
//      int64_t linear_index = *pindex;
//      do {
//        info.data[offset] += *pvalue;
//        pindex++;
//        pvalue++;
//      } while (pindex != sorted_idx_end && *pindex == linear_index);
//      return 0;  // discarded
//    }
/*
    pvalue = psrc + orig_idx;
    for (int64_t e = 0; e < elementsBefore; ++e) {
      int64_t eoffset = beforeIndex[e];
      if (accumulate) {
        info.data[offset + eoffset] += *pvalue;
      } else {
        info.data[offset + eoffset] = *pvalue;
      }
      pvalue += elementsBefore;
    }

    pvalue = psrc + orig_idx * elementsAfter;
    for (int64_t e = 0; e < elementsAfter; ++e) {
      int64_t eoffset = afterIndex[e];
      if (accumulate) {
        info.data[offset + eoffset] += *pvalue;
      } else {
        info.data[offset + eoffset] = *pvalue;
      }
      pvalue++;
    }
*/

//    assert( < dstCopyDimSize);
//    int64_t nn = n / lindexStride;
//    nn = orig_idx_beg[n / lindexStride];
//    const int64_t srcIdx = orig_idx_beg[headIdx] + (lindex % lindexStride);


/*
    int64_t no = orig_idx_beg[n / lindexStride] * lindexStride + (n % lindexStride);
    const int64_t lindex = plindex[no];
    const int64_t headIdx = lindex / headStride;
    const int64_t shPos = lindex % headStride;
    const int64_t offset = headIdx * headStride + shPos;

    printf("%lld %lld %lld %lld %lld %lld %lld\n", n , no, lindex, headStride, headIdx, shPos, offset);



    const T value = p_source[no];
    T* address = info.data + offset;
//    info.data[lindex] += value;

*/

//__device__ __forceinline__ T operator()(int64_t& lindex) {
//    const T* pvalue = p_source + orig_idx;
//    const int64_t offset = indexToOffset<T, int64_t>(info, sorted_idx);

//printf("elementsBefore %lld elementsAfter %lld s=%lld o=%lld\n",
//    elementsBefore, elementsAfter, sorted_idx, orig_idx);

//    if (elementsBefore == 0 && elementsAfter == 0) {
// all dimensions indexed
//      int64_t* pindex = &sorted_idx;
//      int64_t linear_index = *pindex;
//      do {
//        info.data[offset] += *pvalue;
//        pindex++;
//        pvalue++;
//      } while (pindex != sorted_idx_end && *pindex == linear_index);
//      return 0;  // discarded
//    }


//    pvalue = psrc + orig_idx;
//    for (int64_t e = 0; e < elementsBefore; ++e) {
//      int64_t eoffset = beforeIndex[e];
//      if (accumulate) {
//        info.data[offset + eoffset] += *pvalue;
//      } else {
//        info.data[offset + eoffset] = *pvalue;
//      }
//      pvalue += elementsBefore;
//    }
//
//    pvalue = psrc + orig_idx * elementsAfter;
//    for (int64_t e = 0; e < elementsAfter; ++e) {
//      int64_t eoffset = afterIndex[e];
//      if (accumulate) {
//        info.data[offset + eoffset] += *pvalue;
//      } else {
//        info.data[offset + eoffset] = *pvalue;
//      }
//      pvalue++;
//    }


