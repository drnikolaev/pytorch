#include "hip/hip_runtime.h"
// Indexing tensors by by tensors
//
// This corresponds to "advanced indexing" in NumPy. The two operations are:
//
//  index(Tensor self, indices) -> Tensor
//  index_put_(Tensor self, indices, value, accumulate=false)
//
// The index is a TensorList containg kLong or kByte tensors or nulls. Byte
// tensors (boolean masks) are expanded to long tensors via nonzero(). Null
// tensors signify that the dimension is not indexed.
//
// All indexes are broadcast together and iterated as *one*. From NumPy:
//
// result[i_1, ..., i_M] == x[ind_1[i_1, ..., i_M], ind_2[i_1, ..., i_M],
//                           ..., ind_N[i_1, ..., i_M]]
//
// Note 1: ByteTensors expand to index as many dimensions as there are in the
// mask.
//
// Note 2: The behavior is more complicated when the index tensors are not all
// adjacent (e.g. x[[0, 1], :, [2, 3]]). In this case, self and the index
// tensors are transposed to the front: x.transpose(1, 2)[[0, 1], [2, 3]]
//
// The code contains two implementations of indexing. The more efficient
// implementation treats indexing like an elementwise operation over the
// tensors `result`, `x`, `ind_1`, `ind_2`, etc. This implementation does
// not work for index_put_ with accumulate=True. The other implementation
// combines the indexed tensors into a single linear index that is used
// with Tensor.put_. This is used for index_put_ with accumulate=True.
//
// The more efficient implementation takes the following steps for the
// above operation:
//
// 1) Broadcast ind_1, ind_2, ind_3 together to a common shape
// 2) Record x.stride(i) for each indexed dimension `i`
// 3) Replace the indexed subspace of `x` with the shape of the corresponding
//    subspace of `result` but with stride 0
// 4) Add dimensions of size 1 to the index tensors (ind_1, ind_2, etc.) so
//    that their shape is compatible with the result shape
//
// The CPU or CUDA kernel then computes element-wise over the broadcasted
// and restrided result, x, ind_1,  ind_2, etc.:
//
//   result[...] = *(&x[...] +
//                   ind_1[...] * x.stride(1) +
//                   ind_2[...] * x.stride(2) +
//                   ...)
//
// where & and * represent the C-style address-of and indirection operations.

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/ExpandUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <c10/core/ScalarType.h>

#include <ATen/native/Indexing.h>
#include <ATen/NativeFunctions.h>
#include <ATen/LegacyTHFunctions.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <torch/csrc/utils/tensor_flatten.h>

#include <ATen/cpu/vec256/vec256.h>
#include "../../../../../c10/core/DeviceType.h"

namespace at { namespace native {

[[noreturn]]
static void invalid_mask(const Tensor & self, int64_t idx, const Tensor & mask, int64_t maskIdx) {
  std::stringstream ss;
  ss << "The shape of the mask " << mask.sizes() << " at index " << maskIdx;
  ss << " does not match the shape of the indexed tensor " << self.sizes();
  ss << " at index " << idx;
  AT_INDEX_ERROR(ss.str());
}

static void checkIndexTensorTypes(TensorList indices) {
  for (auto& tensor : indices) {
    if (tensor.defined()) {
      auto scalarType = tensor.scalar_type();
      if (scalarType != kLong && scalarType != kByte) {
        AT_INDEX_ERROR("tensors used as indices must be long or byte tensors");
      }
    }
  }
}

static std::vector<Tensor> expandByteTensors(const Tensor & self, TensorList indices) {
  // Expands byte tensors (masks) into the equivalent indexing by LongTensors
  std::vector<Tensor> result;
  for (auto & index : indices) {
    if (index.scalar_type() == kByte) {
      // The sizes of the ByteTensor mask must match the sizes of the
      // corresponding dimensions in self
      for (int64_t j = 0; j < index.dim(); j++) {
        int64_t srcIdx = result.size() + j;
        if (index.size(j) != self.size(srcIdx)) {
          invalid_mask(self, srcIdx, index, j);
        }
      }
      // Replace with nonzeros
      auto nonzero = index.nonzero();
      auto special_empty = false;
      for (int64_t j = 0; j < index.dim(); j++) {
        if (special_empty) {
          // We can't call select on an empty tensor so we just create an empty
          // tensor.
          result.emplace_back(at::empty({0}, nonzero.options()));
        } else {
          result.emplace_back(nonzero.select(1, j));
        }
      }
    } else {
      result.emplace_back(index);
    }
  }
  return result;
}

static bool hasContiguousSubspace(TensorList tl) {
  // true if all the non-null tensors are adjacent
  auto isDefined = [](const Tensor & tensor){ return tensor.defined(); };
  auto isNull = [](const Tensor & tensor){ return !tensor.defined(); };
  auto start = std::find_if(tl.begin(), tl.end(), isDefined);
  auto stop = std::find_if(tl.rbegin(), tl.rend(), isDefined);
  auto it = std::find_if(start, stop.base(), isNull);
  return it == stop.base();
}

// Transposes the tensor and indices together so that all the non-null indices
// index the first k dimensions of the tensor. Returns the transposed tensor
// and the reordered indices. For example:
//  transposeToFront(tensor, {nullptr, a, nullptr, b})
// returns
//  tensor.permute([1, 3, 0, 2]), {a, b, nullptr, nullptr}
static std::tuple<Tensor, std::vector<Tensor>>
transposeToFront(Tensor self, TensorList indices) {
  std::vector<int64_t> dims;
  std::vector<Tensor> transposedIndices;
  dims.reserve(self.dim());
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back(indices[i]);
    }
  }
  for (int64_t i = 0; i < self.dim(); i++) {
    if (!indices[i].defined()) {
      dims.push_back(i);
      transposedIndices.emplace_back();
    }
  }
  return std::make_tuple(self.permute(dims), std::move(transposedIndices));
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contigous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

// Unsqueezes src `before` times at the front and `after` times at the end
//static Tensor unsqueezeN(const Tensor & src, int64_t before, int64_t after) {
//  auto srcSizes = src.sizes();
//  auto nDim = src.dim();
//  std::vector<int64_t> sizes(nDim + before + after, 1);
//  for (int64_t i = 0; i < nDim; i++) {
//    sizes[i + before] = srcSizes[i];
//  }
//  return src.view(sizes);
//}

static std::tuple<Tensor, Tensor, Tensor, Tensor> makeLinearIndex(Tensor self, TensorList orig) {
  checkIndexTensorTypes(orig);
  // first expand ByteTensor (boolean masks) into 1 or more LongTensors
  auto indices = expandByteTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices) = transposeToFront(self, indices);
  }

  auto strides = computeLinearStride(self);

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1;
  for (int64_t i = 0; i < self.dim(); i++) {
    if (indices[i].defined()) {
      if (linearIndex.defined()) {
        linearIndex += indices[i].remainder(self.size(i)) * strides[i];
      } else {
        linearIndex = indices[i].remainder(self.size(i)) * strides[i];
      }
    }
    else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= self.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= self.size(i);
    }
  }
  // Compute the linear indices for the parts of the tensor not being indexed
  // ...and not being sorted
  Tensor beforeIndex;
  if (emptyBefore > 0) {
    beforeIndex = at::arange(0, nElemBefore, self.options().dtype(kLong)) * strides[emptyBefore - 1];
    //      index = index.view(src.sizes().slice(0, emptyBefore));
    //      beforeIndex = unsqueezeN(index, 0, linearIndex.dim() + emptyAfter);
  }
  Tensor afterIndex;
  if (emptyAfter > 0) {
    afterIndex = at::arange(0, nElemAfter, self.options().dtype(kLong));
    //      index = index.view(src.sizes().slice(src.dim() - emptyAfter, emptyAfter));
    //      afterIndex = unsqueezeN(index, linearIndex.dim() + emptyBefore, 0);
  }
  return std::make_tuple(self, linearIndex, beforeIndex, afterIndex);
}

template <typename T, typename IndexType>
__device__ __forceinline__ IndexType indexToOffset(
    const at::cuda::detail::TensorInfo<T, IndexType>& info, IndexType linearIndex) {
  IndexType offset(0);
  for (int i = info.dims - 1; i > 0; --i) {
    offset += (linearIndex % info.sizes[i]) * info.strides[i];
    linearIndex /= info.sizes[i];
  }
  return offset + linearIndex * info.strides[0];
}


template <typename T>
struct TensorPutOp : thrust::unary_function<int64_t, T> {

  TensorPutOp(at::cuda::detail::TensorInfo<T, int64_t> info, bool accumulate,
      const T* psrc, const int64_t* sorted_idx_beg, const int64_t* sorted_idx_end,
      const int64_t* beforeIndex, int64_t elementsBefore,
      const int64_t* afterIndex, int64_t elementsAfter)
    : info(info), accumulate(accumulate), psrc(psrc),
      sorted_idx_beg(sorted_idx_beg), sorted_idx_end(sorted_idx_end),
      beforeIndex(beforeIndex), elementsBefore(elementsBefore),
      afterIndex(afterIndex), elementsAfter(elementsAfter) {}

//  __device__ __forceinline__ T operator()(int64_t& e) {
//
//    pvalue = psrc + orig_idx;
//    int64_t eoffset = beforeIndex[e];
//    if (accumulate) {
//      info.data[offset + eoffset] += *pvalue;
//    } else {
//      info.data[offset + eoffset] = *pvalue;
//    }
//    pvalue += elementsBefore;
//
//
//    return 0;  // discarded
//  }

  __device__ __forceinline__ T operator()(int64_t& sorted_idx, int64_t& orig_idx) {

    const T* pvalue = psrc + orig_idx;
    const int64_t offset = indexToOffset<T, int64_t>(info, sorted_idx);

    if (elementsBefore == 0 && elementsAfter == 0) {
      // all dimensions indexed
      int64_t* pindex = &sorted_idx;
      int64_t linear_index = *pindex;
      do {
        info.data[offset] += *pvalue;
        pindex++;
        pvalue++;
      } while (pindex != sorted_idx_end && *pindex == linear_index);
      return 0;  // discarded
    }

    pvalue = psrc + orig_idx;
    for (int64_t e = 0; e < elementsBefore; ++e) {
      int64_t eoffset = beforeIndex[e];
      if (accumulate) {
        info.data[offset + eoffset] += *pvalue;
      } else {
        info.data[offset + eoffset] = *pvalue;
      }
      pvalue += elementsBefore;
    }

    pvalue = psrc + orig_idx * elementsAfter;
    for (int64_t e = 0; e < elementsAfter; ++e) {
      int64_t eoffset = afterIndex[e];
      if (accumulate) {
        info.data[offset + eoffset] += *pvalue;
      } else {
        info.data[offset + eoffset] = *pvalue;
      }
      pvalue++;
    }
    return 0;  // discarded
  }

  const at::cuda::detail::TensorInfo<T, int64_t> info;
  const bool accumulate;
  const T* psrc;
  const int64_t* sorted_idx_beg;
  const int64_t* sorted_idx_end;
  const int64_t* beforeIndex;
  const int64_t elementsBefore;
  const int64_t* afterIndex;
  const int64_t elementsAfter;
};

Tensor & xput_cuda_(Tensor & self, const Tensor & index, const Tensor & source, bool accumulate,
    const Tensor & beforeIndex, const Tensor & afterIndex) {

  auto sorted_index = index.clone();
  auto orig_index = at::empty_like(index);
  int64_t dstSize = self.numel();
  int64_t idxSize = index.numel();

  auto orig_index_iter = thrust::device_ptr<int64_t>(orig_index.data<int64_t>());
  auto sorted_index_iter = thrust::device_ptr<int64_t>(sorted_index.data<int64_t>());

  const int64_t* sorted_idx_beg = sorted_index.data<int64_t>();
  const int64_t* sorted_idx_end = sorted_idx_beg + idxSize;
  const int64_t* beforeIndexPtr = beforeIndex.defined() ? beforeIndex.data<int64_t>() : nullptr;
  const int64_t  elementsBefore = beforeIndex.defined() ? beforeIndex.numel() : 0L;
  const int64_t* afterIndexPtr  = afterIndex.defined()  ? afterIndex.data<int64_t>() : nullptr;
  const int64_t  elementsAfter  = afterIndex.defined()  ? afterIndex.numel() : 0L;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "xput_cuda_", [&] {
    auto src_iter = thrust::device_ptr<scalar_t>(source.data<scalar_t>());
    auto dst_iter = thrust::make_discard_iterator(); // we directly write to info.data
    auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
    TensorPutOp<scalar_t> putOp(self_info, accumulate, source.data<scalar_t>(),
        sorted_idx_beg, sorted_idx_end, beforeIndexPtr, elementsBefore, afterIndexPtr, elementsAfter);

    thrust::sequence(orig_index_iter, orig_index_iter + idxSize);

    thrust::sort_by_key(
        policy,
        sorted_index_iter, sorted_index_iter + idxSize, orig_index_iter, ThrustLTOp<int64_t>());

    // TODO: for_each
    thrust::transform(
        policy,
        sorted_index_iter,
        sorted_index_iter + idxSize,
        orig_index_iter,
        dst_iter, putOp);

//    auto first = thrust::make_zip_iterator(thrust::make_tuple(
//        sorted_index_iter, orig_index_iter));
//    auto last = thrust::make_zip_iterator(thrust::make_tuple(
//        sorted_index_iter + idxSize, orig_index_iter + idxSize))    ;
//    thrust::for_each(policy, first, last, putOp);

  });

  return self;
}


//long long mtotal = 0;
//long long xtotal = 0;
//long pcnt = 0;

Tensor & index_put_cuda_(Tensor & self, TensorList indices, const Tensor & value, bool accumulate) {
  if (indices.size() > (size_t)self.dim()) {
    AT_INDEX_ERROR("too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }

  Tensor src, linearIndex, expandedValue;
  Tensor beforeIndex, afterIndex;

//  ++pcnt;
//  auto start = std::chrono::high_resolution_clock::now();

  std::tie(src, linearIndex, beforeIndex, afterIndex) = makeLinearIndex(self, indices);

//  auto finish = std::chrono::high_resolution_clock::now();
//
//  mtotal += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//  if (pcnt % 100 == 0) {
//    std::cout << "makeLinearIndex: "
//              << mtotal/pcnt/1000
//              << "us" << std::endl;;
//  }


//  std::tie(expandedValue) = expand_inplace(linearIndex, value);

//  start = std::chrono::high_resolution_clock::now();
  Tensor & ret = src.xput_(linearIndex, value, accumulate, beforeIndex, afterIndex);
//  finish = std::chrono::high_resolution_clock::now();
//
//  xtotal += std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
//  if (pcnt % 100 == 0) {
//    std::cout << "xput: "
//              << xtotal/pcnt/1000
//              << "us" << std::endl;;
//  }

  return ret;
//  return src.xput_(linearIndex, value, accumulate);
}

}}
