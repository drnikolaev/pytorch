#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/Exceptions.h>

#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

template<typename T, typename accT = T>
struct LinspaceOp {
  __host__ __device__ LinspaceOp(accT start, accT step):
    start_(start), step_(step) { }
  __device__ __forceinline__ T operator()(ptrdiff_t index) {
    accT increment = step_ * static_cast<accT>(index);
    accT value = start_ + increment;
    return static_cast<T>(value);
  }

  const accT start_, step_;
};

Tensor& linspace_cuda_out(Tensor& result, Scalar start, Scalar end, int64_t steps) {
  AT_CHECK(steps >= 0, "number of steps must be non-negative");

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  Tensor r = result.is_contiguous() ? result : result.contiguous();

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(start);
  } else {
    AT_DISPATCH_FLOATING_TYPES(r.type(), "linspace", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);
      LinspaceOp<scalar_t> linspace_method(scalar_start, step);
      thrust::device_ptr<scalar_t> data_(r.data<scalar_t>());
      thrust::tabulate(data_, data_ + steps, linspace_method);
    });
  }

  if (!result.is_contiguous()) {
    result.copy_(r);
  }
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

}} // namespace at::native
